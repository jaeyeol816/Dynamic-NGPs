#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   openxr_hmd.cu
 *  @author Thomas Müller & Ingo Esser & Robert Menzel, NVIDIA
 *  @brief  Wrapper around the OpenXR API, providing access to
 *          per-eye framebuffers, lens parameters, visible area,
 *          view, hand, and eye poses, as well as controller inputs.
 */

#define NOMINMAX

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/openxr_hmd.h>
#include <neural-graphics-primitives/render_buffer.h>

#include <openxr/openxr_reflection.h>

#include <fmt/format.h>

#include <imgui/imgui.h>

#include <tinylogger/tinylogger.h>

#include <tiny-cuda-nn/common.h>

#include <string>
#include <vector>

#ifdef __GNUC__
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wmissing-field-initializers" //TODO: XR struct are uninitiaized apart from their type
#endif

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

// function XrEnumStr turns enum into string for printing
// uses expansion macro and data provided in openxr_reflection.h
#define XR_ENUM_CASE_STR(name, val) \
	case name:                      \
		return #name;
#define XR_ENUM_STR(enum_type)                                                     \
	constexpr const char* XrEnumStr(enum_type e) {                                 \
		switch (e) {                                                               \
			XR_LIST_ENUM_##enum_type(XR_ENUM_CASE_STR) default : return "Unknown"; \
		}                                                                          \
	}

XR_ENUM_STR(XrViewConfigurationType)
XR_ENUM_STR(XrEnvironmentBlendMode)
XR_ENUM_STR(XrReferenceSpaceType)
XR_ENUM_STR(XrStructureType)
XR_ENUM_STR(XrSessionState)

/// Checks the result of a xrXXXXXX call and throws an error on failure
#define XR_CHECK_THROW(x)                                                                                   \
	do {                                                                                                              \
		XrResult result = x;                                                                                          \
		if (XR_FAILED(result)) {                                                                                      \
			char buffer[XR_MAX_RESULT_STRING_SIZE];                                                                   \
			XrResult result_to_string_result = xrResultToString(m_instance, result, buffer);                            \
			if (XR_FAILED(result_to_string_result)) {                                                                 \
				throw std::runtime_error{std::string(FILE_LINE " " #x " failed, but could not obtain error string")}; \
			} else {                                                                                                  \
				throw std::runtime_error{std::string(FILE_LINE " " #x " failed with error ") + buffer};               \
			}                                                                                                         \
		}                                                                                                             \
	} while(0)

OpenXRHMD::Swapchain::Swapchain(XrSwapchainCreateInfo& rgba_create_info, XrSwapchainCreateInfo& depth_create_info, XrSession& session, XrInstance& m_instance) {
	ScopeGuard cleanup_guard{[&]() { clear(); }};

	XR_CHECK_THROW(xrCreateSwapchain(session, &rgba_create_info, &handle));

	width = rgba_create_info.width;
	height = rgba_create_info.height;

	{
		uint32_t size;
		XR_CHECK_THROW(xrEnumerateSwapchainImages(handle, 0, &size, nullptr));

		images_gl.resize(size, {XR_TYPE_SWAPCHAIN_IMAGE_OPENGL_KHR});
		XR_CHECK_THROW(xrEnumerateSwapchainImages(handle, size, &size, (XrSwapchainImageBaseHeader*)images_gl.data()));

		// One framebuffer per swapchain image
		framebuffers_gl.resize(size);
	}

	if (depth_create_info.format != 0) {
		XR_CHECK_THROW(xrCreateSwapchain(session, &depth_create_info, &depth_handle));

		uint32_t depth_size;
		XR_CHECK_THROW(xrEnumerateSwapchainImages(depth_handle, 0, &depth_size, nullptr));

		depth_images_gl.resize(depth_size, {XR_TYPE_SWAPCHAIN_IMAGE_OPENGL_KHR});
		XR_CHECK_THROW(xrEnumerateSwapchainImages(depth_handle, depth_size, &depth_size, (XrSwapchainImageBaseHeader*)depth_images_gl.data()));

		// We might have a different number of depth swapchain images as we have framebuffers,
		// so we will need to bind an acquired depth image to the current framebuffer on the
		// fly later on.
	}

	glGenFramebuffers(framebuffers_gl.size(), framebuffers_gl.data());

	cleanup_guard.disarm();
}

OpenXRHMD::Swapchain::~Swapchain() {
	clear();
}

void OpenXRHMD::Swapchain::clear() {
	if (!framebuffers_gl.empty()) {
		glDeleteFramebuffers(framebuffers_gl.size(), framebuffers_gl.data());
	}

	if (depth_handle != XR_NULL_HANDLE) {
		xrDestroySwapchain(depth_handle);
		depth_handle = XR_NULL_HANDLE;
	}

	if (handle != XR_NULL_HANDLE) {
		xrDestroySwapchain(handle);
		handle = XR_NULL_HANDLE;
	}
}

#if defined(XR_USE_PLATFORM_WIN32)
OpenXRHMD::OpenXRHMD(HDC hdc, HGLRC hglrc) {
#elif defined(XR_USE_PLATFORM_XLIB)
OpenXRHMD::OpenXRHMD(Display* xDisplay, uint32_t visualid, GLXFBConfig glxFBConfig, GLXDrawable glxDrawable, GLXContext glxContext) {
#elif defined(XR_USE_PLATFORM_WAYLAND)
OpenXRHMD::OpenXRHMD(wl_display* display) {
#endif
	ScopeGuard cleanup_guard{[&]() { clear(); }};

	init_create_xr_instance();
	init_get_xr_system();
	init_configure_xr_views();
	init_check_for_xr_blend_mode();
#if defined(XR_USE_PLATFORM_WIN32)
	init_open_gl(hdc, hglrc);
#elif defined(XR_USE_PLATFORM_XLIB)
	init_open_gl(xDisplay, visualid, glxFBConfig, glxDrawable, glxContext);
#elif defined(XR_USE_PLATFORM_WAYLAND)
	init_open_gl(display);
#endif
	init_xr_session();
	init_xr_actions();
	init_xr_spaces();
	init_xr_swapchain_open_gl();
	init_open_gl_shaders();

	cleanup_guard.disarm();
	tlog::success() << "Initialized OpenXR for " << m_system_properties.systemName;
	// tlog::success() << " "
	// 	<< " depth=" << (m_supports_composition_layer_depth ? "true" : "false")
	// 	<< " mask=" << (m_supports_hidden_area_mask ? "true" : "false")
	// 	<< " eye=" << (m_supports_eye_tracking ? "true" : "false")
	// 	;
}

OpenXRHMD::~OpenXRHMD() {
	clear();
}

void OpenXRHMD::clear() {
	auto xr_destroy = [&](auto& handle, auto destroy_fun) {
		if (handle != XR_NULL_HANDLE) {
			destroy_fun(handle);
			handle = XR_NULL_HANDLE;
		}
	};

	xr_destroy(m_pose_action, xrDestroyAction);
	xr_destroy(m_thumbstick_actions[0], xrDestroyAction);
	xr_destroy(m_thumbstick_actions[1], xrDestroyAction);
	xr_destroy(m_press_action, xrDestroyAction);
	xr_destroy(m_grab_action, xrDestroyAction);

	xr_destroy(m_action_set, xrDestroyActionSet);

	m_swapchains.clear();
	xr_destroy(m_space, xrDestroySpace);
	xr_destroy(m_session, xrDestroySession);
	xr_destroy(m_instance, xrDestroyInstance);
}

void OpenXRHMD::init_create_xr_instance() {
	std::vector<const char*> layers = {};
	std::vector<const char*> extensions = {
		XR_KHR_OPENGL_ENABLE_EXTENSION_NAME,
	};

	auto print_extension_properties = [](const char* layer_name) {
		uint32_t size;
		xrEnumerateInstanceExtensionProperties(layer_name, 0, &size, nullptr);
		std::vector<XrExtensionProperties> props(size, {XR_TYPE_EXTENSION_PROPERTIES});
		xrEnumerateInstanceExtensionProperties(layer_name, size, &size, props.data());
		tlog::info() << fmt::format("Extensions ({}):", props.size());
		for (XrExtensionProperties extension : props) {
			tlog::info() << fmt::format("\t{} (Version {})", extension.extensionName, extension.extensionVersion);
		}
	};

	uint32_t size;
	xrEnumerateApiLayerProperties(0, &size, nullptr);
	m_api_layer_properties.clear();
	m_api_layer_properties.resize(size, {XR_TYPE_API_LAYER_PROPERTIES});
	xrEnumerateApiLayerProperties(size, &size, m_api_layer_properties.data());

	if (m_print_api_layers) {
		tlog::info() << fmt::format("API Layers ({}):", m_api_layer_properties.size());
		for (auto p : m_api_layer_properties) {
			tlog::info() << fmt::format(
				"{} (v {}.{}.{}, {}) {}",
				p.layerName,
				XR_VERSION_MAJOR(p.specVersion),
				XR_VERSION_MINOR(p.specVersion),
				XR_VERSION_PATCH(p.specVersion),
				p.layerVersion,
				p.description
			);
			print_extension_properties(p.layerName);
		}
	}

	if (layers.size() != 0) {
		for (const auto& e : layers) {
			bool found = false;
			for (XrApiLayerProperties layer : m_api_layer_properties) {
				if (strcmp(e, layer.layerName) == 0) {
					found = true;
					break;
				}
			}

			if (!found) {
				throw std::runtime_error{fmt::format("OpenXR API layer {} not found", e)};
			}
		}
	}

	xrEnumerateInstanceExtensionProperties(nullptr, 0, &size, nullptr);
	m_instance_extension_properties.clear();
	m_instance_extension_properties.resize(size, {XR_TYPE_EXTENSION_PROPERTIES});
	xrEnumerateInstanceExtensionProperties(nullptr, size, &size, m_instance_extension_properties.data());

	if (m_print_extensions) {
		tlog::info() << fmt::format("Instance extensions ({}):", m_instance_extension_properties.size());
		for (XrExtensionProperties extension : m_instance_extension_properties) {
			tlog::info() << fmt::format("\t{} (Version {})", extension.extensionName, extension.extensionVersion);
		}
	}

	auto has_extension = [&](const char* e) {
		for (XrExtensionProperties extension : m_instance_extension_properties) {
			if (strcmp(e, extension.extensionName) == 0) {
				return true;
			}
		}

		return false;
	};

	for (const auto& e : extensions) {
		if (!has_extension(e)) {
			throw std::runtime_error{fmt::format("Required OpenXR extension {} not found", e)};
		}
	}

	auto add_extension_if_supported = [&](const char* extension) {
		if (has_extension(extension)) {
			extensions.emplace_back(extension);
			return true;
		}

		return false;
	};

	if (add_extension_if_supported(XR_KHR_COMPOSITION_LAYER_DEPTH_EXTENSION_NAME)) {
		m_supports_composition_layer_depth = true;
	}

	if (add_extension_if_supported(XR_KHR_VISIBILITY_MASK_EXTENSION_NAME)) {
		m_supports_hidden_area_mask = true;
	}

	if (add_extension_if_supported(XR_EXT_EYE_GAZE_INTERACTION_EXTENSION_NAME)) {
		m_supports_eye_tracking = true;
	}

	XrInstanceCreateInfo instance_create_info = {XR_TYPE_INSTANCE_CREATE_INFO};
	instance_create_info.applicationInfo = {};
	strncpy(instance_create_info.applicationInfo.applicationName, "Instant Neural Graphics Primitives v" NGP_VERSION, XR_MAX_APPLICATION_NAME_SIZE);
	instance_create_info.applicationInfo.applicationVersion = 1;
	strncpy(instance_create_info.applicationInfo.engineName, "Instant Neural Graphics Primitives v" NGP_VERSION, XR_MAX_ENGINE_NAME_SIZE);
	instance_create_info.applicationInfo.engineVersion = 1;
	instance_create_info.applicationInfo.apiVersion = XR_CURRENT_API_VERSION;
	instance_create_info.enabledExtensionCount = (uint32_t)extensions.size();
	instance_create_info.enabledExtensionNames = extensions.data();
	instance_create_info.enabledApiLayerCount = (uint32_t)layers.size();
	instance_create_info.enabledApiLayerNames = layers.data();

	if (XR_FAILED(xrCreateInstance(&instance_create_info, &m_instance))) {
		throw std::runtime_error{"Failed to create OpenXR instance"};
	}

	XR_CHECK_THROW(xrGetInstanceProperties(m_instance, &m_instance_properties));
	if (m_print_instance_properties) {
		tlog::info() << "Instance Properties";
		tlog::info() << fmt::format("\t        runtime name: '{}'", m_instance_properties.runtimeName);
		const auto& v = m_instance_properties.runtimeVersion;
		tlog::info() << fmt::format(
			"\t     runtime version: {}.{}.{}",
			XR_VERSION_MAJOR(v),
			XR_VERSION_MINOR(v),
			XR_VERSION_PATCH(v)
		);
	}
}

void OpenXRHMD::init_get_xr_system() {
	XrSystemGetInfo system_get_info = {XR_TYPE_SYSTEM_GET_INFO, nullptr, XR_FORM_FACTOR_HEAD_MOUNTED_DISPLAY};
	XR_CHECK_THROW(xrGetSystem(m_instance, &system_get_info, &m_system_id));

	XR_CHECK_THROW(xrGetSystemProperties(m_instance, m_system_id, &m_system_properties));
	if (m_print_system_properties) {
		tlog::info() << "System Properties";
		tlog::info() << fmt::format("\t                name: '{}'", m_system_properties.systemName);
		tlog::info() << fmt::format("\t            vendorId: {:#x}", m_system_properties.vendorId);
		tlog::info() << fmt::format("\t            systemId: {:#x}", m_system_properties.systemId);
		tlog::info() << fmt::format("\t     max layer count: {}", m_system_properties.graphicsProperties.maxLayerCount);
		tlog::info() << fmt::format("\t       max img width: {}", m_system_properties.graphicsProperties.maxSwapchainImageWidth);
		tlog::info() << fmt::format("\t      max img height: {}", m_system_properties.graphicsProperties.maxSwapchainImageHeight);
		tlog::info() << fmt::format("\torientation tracking: {}", m_system_properties.trackingProperties.orientationTracking ? "YES" : "NO");
		tlog::info() << fmt::format("\t   position tracking: {}", m_system_properties.trackingProperties.orientationTracking ? "YES" : "NO");
	}
}

void OpenXRHMD::init_configure_xr_views() {
	uint32_t size;
	XR_CHECK_THROW(xrEnumerateViewConfigurations(m_instance, m_system_id, 0, &size, nullptr));
	std::vector<XrViewConfigurationType> view_config_types(size);
	XR_CHECK_THROW(xrEnumerateViewConfigurations(m_instance, m_system_id, size, &size, view_config_types.data()));

	if (m_print_view_configuration_types) {
		tlog::info() << fmt::format("View Configuration Types ({}):", view_config_types.size());
		for (const auto& t : view_config_types) {
			tlog::info() << fmt::format("\t{}", XrEnumStr(t));
		}
	}

	// view configurations we support, in descending preference
	const std::vector<XrViewConfigurationType> preferred_view_config_types = {
		//XR_VIEW_CONFIGURATION_TYPE_PRIMARY_QUAD_VARJO,
		XR_VIEW_CONFIGURATION_TYPE_PRIMARY_STEREO
	};

	bool found = false;
	for (const auto& p : preferred_view_config_types) {
		for (const auto& t : view_config_types) {
			if (p == t) {
				found = true;
				m_view_configuration_type = t;
			}
		}
	}

	if (!found) {
		throw std::runtime_error{"Could not find a suitable OpenXR view configuration type"};
	}

	// get view configuration properties
	XR_CHECK_THROW(xrGetViewConfigurationProperties(m_instance, m_system_id, m_view_configuration_type, &m_view_configuration_properties));
	if (m_print_view_configuration_properties) {
		tlog::info() << "View Configuration Properties:";
		tlog::info() << fmt::format("\t         Type: {}", XrEnumStr(m_view_configuration_type));
		tlog::info() << fmt::format("\t         FOV Mutable: {}", m_view_configuration_properties.fovMutable ? "YES" : "NO");
	}

	// enumerate view configuration views
	XR_CHECK_THROW(xrEnumerateViewConfigurationViews(m_instance, m_system_id, m_view_configuration_type, 0, &size, nullptr));
	m_view_configuration_views.clear();
	m_view_configuration_views.resize(size, {XR_TYPE_VIEW_CONFIGURATION_VIEW});
	XR_CHECK_THROW(xrEnumerateViewConfigurationViews(
		m_instance,
		m_system_id,
		m_view_configuration_type,
		size,
		&size,
		m_view_configuration_views.data()
	));

	if (m_print_view_configuration_view) {
		tlog::info() << "View Configuration Views, Width x Height x Samples";
		for (size_t i = 0; i < m_view_configuration_views.size(); ++i) {
			const auto& view = m_view_configuration_views[i];
			tlog::info() << fmt::format(
				"\tView {}\tRecommended: {}x{}x{}  Max: {}x{}x{}",
				i,
				view.recommendedImageRectWidth,
				view.recommendedImageRectHeight,
				view.recommendedSwapchainSampleCount,
				view.maxImageRectWidth,
				view.maxImageRectHeight,
				view.maxSwapchainSampleCount
			);
		}
	}
}

void OpenXRHMD::init_check_for_xr_blend_mode() {
	// enumerate environment blend modes
	uint32_t size;
	XR_CHECK_THROW(xrEnumerateEnvironmentBlendModes(m_instance, m_system_id, m_view_configuration_type, 0, &size, nullptr));
	std::vector<XrEnvironmentBlendMode> supported_blend_modes(size);
	XR_CHECK_THROW(xrEnumerateEnvironmentBlendModes(
		m_instance,
		m_system_id,
		m_view_configuration_type,
		size,
		&size,
		supported_blend_modes.data()
	));

	if (supported_blend_modes.empty()) {
		throw std::runtime_error{"No OpenXR environment blend modes found"};
	}

	std::sort(std::begin(supported_blend_modes), std::end(supported_blend_modes));
	if (m_print_environment_blend_modes) {
		tlog::info() << fmt::format("Environment Blend Modes ({}):", supported_blend_modes.size());
	}

	m_supported_environment_blend_modes.resize(supported_blend_modes.size());
	m_supported_environment_blend_modes_imgui_string.clear();
	for (size_t i = 0; i < supported_blend_modes.size(); ++i) {
		if (m_print_environment_blend_modes) {
			tlog::info() << fmt::format("\t{}", XrEnumStr(supported_blend_modes[i]));
		}

		auto b = (EEnvironmentBlendMode)supported_blend_modes[i];
		m_supported_environment_blend_modes[i] = b;

		auto b_str = to_string(b);
		std::copy(std::begin(b_str), std::end(b_str), std::back_inserter(m_supported_environment_blend_modes_imgui_string));
		m_supported_environment_blend_modes_imgui_string.emplace_back('\0');
	}

	m_supported_environment_blend_modes_imgui_string.emplace_back('\0');
	m_environment_blend_mode = m_supported_environment_blend_modes.front();
}

void OpenXRHMD::init_xr_actions() {
	// paths for left (0) and right (1) hands
	XR_CHECK_THROW(xrStringToPath(m_instance, "/user/hand/left", &m_hand_paths[0]));
	XR_CHECK_THROW(xrStringToPath(m_instance, "/user/hand/right", &m_hand_paths[1]));

	// create action set
	XrActionSetCreateInfo action_set_create_info{XR_TYPE_ACTION_SET_CREATE_INFO, nullptr, "actionset", "actionset", 0};
	XR_CHECK_THROW(xrCreateActionSet(m_instance, &action_set_create_info, &m_action_set));

	{
		XrActionCreateInfo action_create_info{
			XR_TYPE_ACTION_CREATE_INFO,
			nullptr,
			"hand_pose",
			XR_ACTION_TYPE_POSE_INPUT,
			(uint32_t)m_hand_paths.size(),
			m_hand_paths.data(),
			"Hand pose"
		};
		XR_CHECK_THROW(xrCreateAction(m_action_set, &action_create_info, &m_pose_action));
	}

	{
		XrActionCreateInfo action_create_info{
			XR_TYPE_ACTION_CREATE_INFO,
			nullptr,
			"thumbstick_left",
			XR_ACTION_TYPE_VECTOR2F_INPUT,
			0,
			nullptr,
			"Left thumbstick"
		};
		XR_CHECK_THROW(xrCreateAction(m_action_set, &action_create_info, &m_thumbstick_actions[0]));
	}

	{
		XrActionCreateInfo action_create_info{
			XR_TYPE_ACTION_CREATE_INFO,
			nullptr,
			"thumbstick_right",
			XR_ACTION_TYPE_VECTOR2F_INPUT,
			0,
			nullptr,
			"Right thumbstick"
		};
		XR_CHECK_THROW(xrCreateAction(m_action_set, &action_create_info, &m_thumbstick_actions[1]));
	}

	{
		XrActionCreateInfo action_create_info{
			XR_TYPE_ACTION_CREATE_INFO,
			nullptr,
			"press",
			XR_ACTION_TYPE_BOOLEAN_INPUT,
			(uint32_t)m_hand_paths.size(),
			m_hand_paths.data(),
			"Press"
		};
		XR_CHECK_THROW(xrCreateAction(m_action_set, &action_create_info, &m_press_action));
	}

	{
		XrActionCreateInfo action_create_info{
			XR_TYPE_ACTION_CREATE_INFO,
			nullptr,
			"grab",
			XR_ACTION_TYPE_FLOAT_INPUT,
			(uint32_t)m_hand_paths.size(),
			m_hand_paths.data(),
			"Grab"
		};
		XR_CHECK_THROW(xrCreateAction(m_action_set, &action_create_info, &m_grab_action));
	}

	auto create_binding = [&](XrAction action, const std::string& binding_path_str) {
		XrPath binding;
		XR_CHECK_THROW(xrStringToPath(m_instance, binding_path_str.c_str(), &binding));
		return XrActionSuggestedBinding{action, binding};
	};

	auto suggest_bindings = [&](const std::string& interaction_profile_path_str, const std::vector<XrActionSuggestedBinding>& bindings) {
		XrPath interaction_profile;
		XR_CHECK_THROW(xrStringToPath(m_instance, interaction_profile_path_str.c_str(), &interaction_profile));
		XrInteractionProfileSuggestedBinding suggested_binding{
			XR_TYPE_INTERACTION_PROFILE_SUGGESTED_BINDING,
			nullptr,
			interaction_profile,
			(uint32_t)bindings.size(),
			bindings.data()
		};
		XR_CHECK_THROW(xrSuggestInteractionProfileBindings(m_instance, &suggested_binding));
	};

	suggest_bindings("/interaction_profiles/khr/simple_controller", {
		create_binding(m_pose_action, "/user/hand/left/input/grip/pose"),
		create_binding(m_pose_action, "/user/hand/right/input/grip/pose"),
	});

	auto suggest_controller_bindings = [&](const std::string& xy, const std::string& press, const std::string& grab, const std::string& squeeze, const std::string& interaction_profile_path_str) {
		std::vector<XrActionSuggestedBinding> bindings = {
			create_binding(m_pose_action, "/user/hand/left/input/grip/pose"),
			create_binding(m_pose_action, "/user/hand/right/input/grip/pose"),
			create_binding(m_thumbstick_actions[0], std::string{"/user/hand/left/input/"} + xy),
			create_binding(m_thumbstick_actions[1], std::string{"/user/hand/right/input/"} + xy),
			create_binding(m_press_action, std::string{"/user/hand/left/input/"} + press),
			create_binding(m_press_action, std::string{"/user/hand/right/input/"} + press),
			create_binding(m_grab_action, std::string{"/user/hand/left/input/"} + grab),
			create_binding(m_grab_action, std::string{"/user/hand/right/input/"} + grab),
		};

		if (!squeeze.empty()) {
			bindings.emplace_back(create_binding(m_grab_action, std::string{"/user/hand/left/input/"} + squeeze));
			bindings.emplace_back(create_binding(m_grab_action, std::string{"/user/hand/right/input/"} + squeeze));
		}

		suggest_bindings(interaction_profile_path_str, bindings);
	};

	suggest_controller_bindings("trackpad",   "select/click",     "trackpad/click", "",                  "/interaction_profiles/google/daydream_controller");
	suggest_controller_bindings("trackpad",   "trackpad/click",   "trigger/click",  "squeeze/click",     "/interaction_profiles/htc/vive_controller");
	suggest_controller_bindings("thumbstick", "thumbstick/click", "trigger/value",  "squeeze/click",     "/interaction_profiles/microsoft/motion_controller");
	suggest_controller_bindings("trackpad",   "trackpad/click",   "trigger/click",  "",                  "/interaction_profiles/oculus/go_controller");
	suggest_controller_bindings("thumbstick", "thumbstick/click", "trigger/value",  "squeeze/value",     "/interaction_profiles/oculus/touch_controller");

	// Valve Index force squeeze is very sensitive and can cause unwanted grabbing. Only permit trigger-grabbing for now.
	suggest_controller_bindings("thumbstick", "thumbstick/click", "trigger/value",  ""/*squeeze/force*/, "/interaction_profiles/valve/index_controller");

	// Xbox controller (currently not functional)
	suggest_bindings("/interaction_profiles/microsoft/xbox_controller", {
		create_binding(m_thumbstick_actions[0], std::string{"/user/gamepad/input/thumbstick_left"}),
		create_binding(m_thumbstick_actions[1], std::string{"/user/gamepad/input/thumbstick_right"}),
	});
}

#if defined(XR_USE_PLATFORM_WIN32)
void OpenXRHMD::init_open_gl(HDC hdc, HGLRC hglrc) {
#elif defined(XR_USE_PLATFORM_XLIB)
void OpenXRHMD::init_open_gl(Display* xDisplay, uint32_t visualid, GLXFBConfig glxFBConfig, GLXDrawable glxDrawable, GLXContext glxContext) {
#elif defined(XR_USE_PLATFORM_WAYLAND)
void OpenXRHMD::init_open_gl(wl_display* display) {
#endif
	// GL graphics requirements
	PFN_xrGetOpenGLGraphicsRequirementsKHR xrGetOpenGLGraphicsRequirementsKHR = nullptr;
	XR_CHECK_THROW(xrGetInstanceProcAddr(
		m_instance,
		"xrGetOpenGLGraphicsRequirementsKHR",
		reinterpret_cast<PFN_xrVoidFunction*>(&xrGetOpenGLGraphicsRequirementsKHR)
	));

	XrGraphicsRequirementsOpenGLKHR graphics_requirements{XR_TYPE_GRAPHICS_REQUIREMENTS_OPENGL_KHR};
	xrGetOpenGLGraphicsRequirementsKHR(m_instance, m_system_id, &graphics_requirements);
	XrVersion min_version = graphics_requirements.minApiVersionSupported;
	GLint major = 0;
	GLint minor = 0;
	glGetIntegerv(GL_MAJOR_VERSION, &major);
	glGetIntegerv(GL_MINOR_VERSION, &minor);
	const XrVersion have_version = XR_MAKE_VERSION(major, minor, 0);

	if (have_version < min_version) {
		tlog::info() << fmt::format(
			"Required OpenGL version: {}.{}, found OpenGL version: {}.{}",
			XR_VERSION_MAJOR(min_version),
			XR_VERSION_MINOR(min_version),
			major,
			minor
		);

		throw std::runtime_error{"Insufficient graphics API support"};
	}

#if defined(XR_USE_PLATFORM_WIN32)
	m_graphics_binding.hDC = hdc;
	m_graphics_binding.hGLRC = hglrc;
#elif defined(XR_USE_PLATFORM_XLIB)
	m_graphics_binding.xDisplay = xDisplay;
	m_graphics_binding.visualid = visualid;
	m_graphics_binding.glxFBConfig = glxFBConfig;
	m_graphics_binding.glxDrawable = glxDrawable;
	m_graphics_binding.glxContext = glxContext;
#elif defined(XR_USE_PLATFORM_WAYLAND)
	m_graphics_binding.display = display;
#endif
}

void OpenXRHMD::init_xr_session() {
	// create session
	XrSessionCreateInfo create_info{
		XR_TYPE_SESSION_CREATE_INFO,
		reinterpret_cast<const XrBaseInStructure*>(&m_graphics_binding),
		0,
		m_system_id
	};

	XR_CHECK_THROW(xrCreateSession(m_instance, &create_info, &m_session));

	// tlog::info() << fmt::format("Created session {}", fmt::ptr(m_session));
}

void OpenXRHMD::init_xr_spaces() {
	// reference space
	uint32_t size;
	XR_CHECK_THROW(xrEnumerateReferenceSpaces(m_session, 0, &size, nullptr));
	m_reference_spaces.clear();
	m_reference_spaces.resize(size);
	XR_CHECK_THROW(xrEnumerateReferenceSpaces(m_session, size, &size, m_reference_spaces.data()));

	if (m_print_reference_spaces) {
		tlog::info() << fmt::format("Reference spaces ({}):", m_reference_spaces.size());
		for (const auto& r : m_reference_spaces) {
			tlog::info() << fmt::format("\t{}", XrEnumStr(r));
		}
	}

	XrReferenceSpaceCreateInfo reference_space_create_info{XR_TYPE_REFERENCE_SPACE_CREATE_INFO};
	reference_space_create_info.referenceSpaceType = XR_REFERENCE_SPACE_TYPE_LOCAL;
	reference_space_create_info.poseInReferenceSpace = XrPosef{};
	reference_space_create_info.poseInReferenceSpace.orientation.w = 1.0f;
	XR_CHECK_THROW(xrCreateReferenceSpace(m_session, &reference_space_create_info, &m_space));
	XR_CHECK_THROW(xrGetReferenceSpaceBoundsRect(m_session, reference_space_create_info.referenceSpaceType, &m_bounds));

	if (m_print_reference_spaces) {
		tlog::info() << fmt::format("Using reference space {}", XrEnumStr(reference_space_create_info.referenceSpaceType));
		tlog::info() << fmt::format("Reference space boundaries: {} x {}", m_bounds.width, m_bounds.height);
	}

	// action space
	XrActionSpaceCreateInfo action_space_create_info{XR_TYPE_ACTION_SPACE_CREATE_INFO};
	action_space_create_info.action = m_pose_action;
	action_space_create_info.poseInActionSpace.orientation.w = 1.0f;
	action_space_create_info.subactionPath = m_hand_paths[0];
	XR_CHECK_THROW(xrCreateActionSpace(m_session, &action_space_create_info, &m_hand_spaces[0]));
	action_space_create_info.subactionPath = m_hand_paths[1];
	XR_CHECK_THROW(xrCreateActionSpace(m_session, &action_space_create_info, &m_hand_spaces[1]));

	// attach action set
	XrSessionActionSetsAttachInfo attach_info{XR_TYPE_SESSION_ACTION_SETS_ATTACH_INFO};
	attach_info.countActionSets = 1;
	attach_info.actionSets = &m_action_set;
	XR_CHECK_THROW(xrAttachSessionActionSets(m_session, &attach_info));
}

void OpenXRHMD::init_xr_swapchain_open_gl() {
	// swap chains
	uint32_t size;
	XR_CHECK_THROW(xrEnumerateSwapchainFormats(m_session, 0, &size, nullptr));
	std::vector<int64_t> swapchain_formats(size);
	XR_CHECK_THROW(xrEnumerateSwapchainFormats(m_session, size, &size, swapchain_formats.data()));

	if (m_print_available_swapchain_formats) {
		tlog::info() << fmt::format("Swapchain formats ({}):", swapchain_formats.size());
		for (const auto& f : swapchain_formats) {
			tlog::info() << fmt::format("\t{:#x}", f);
		}
	}

	auto find_compatible_swapchain_format = [&](const std::vector<int64_t>& candidates) {
		for (auto format : candidates) {
			if (std::find(std::begin(swapchain_formats), std::end(swapchain_formats), format) != std::end(swapchain_formats)) {
				return format;
			}
		}

		throw std::runtime_error{"No compatible OpenXR swapchain format found"};
	};

	m_swapchain_rgba_format = find_compatible_swapchain_format({
		GL_SRGB8_ALPHA8,
		GL_SRGB8,
		GL_RGBA8,
	});

	if (m_supports_composition_layer_depth) {
		m_swapchain_depth_format = find_compatible_swapchain_format({
			GL_DEPTH_COMPONENT32F,
			GL_DEPTH_COMPONENT24,
			GL_DEPTH_COMPONENT16,
		});
	}

	// tlog::info() << fmt::format("Chosen swapchain format: {:#x}", m_swapchain_rgba_format);
	for (const auto& vcv : m_view_configuration_views) {
		XrSwapchainCreateInfo rgba_swapchain_create_info{XR_TYPE_SWAPCHAIN_CREATE_INFO};
		rgba_swapchain_create_info.usageFlags = XR_SWAPCHAIN_USAGE_SAMPLED_BIT | XR_SWAPCHAIN_USAGE_COLOR_ATTACHMENT_BIT;
		rgba_swapchain_create_info.format = m_swapchain_rgba_format;
		rgba_swapchain_create_info.sampleCount = 1;
		rgba_swapchain_create_info.width = vcv.recommendedImageRectWidth;
		rgba_swapchain_create_info.height = vcv.recommendedImageRectHeight;
		rgba_swapchain_create_info.faceCount = 1;
		rgba_swapchain_create_info.arraySize = 1;
		rgba_swapchain_create_info.mipCount = 1;

		XrSwapchainCreateInfo depth_swapchain_create_info = rgba_swapchain_create_info;
		depth_swapchain_create_info.usageFlags = XR_SWAPCHAIN_USAGE_SAMPLED_BIT | XR_SWAPCHAIN_USAGE_DEPTH_STENCIL_ATTACHMENT_BIT;
		depth_swapchain_create_info.format = m_swapchain_depth_format;

		m_swapchains.emplace_back(rgba_swapchain_create_info, depth_swapchain_create_info, m_session, m_instance);
	}
}

void OpenXRHMD::init_open_gl_shaders() {
	// Hidden area mask program
	{
		static const char* shader_vert = R"(#version 140
			in vec2 pos;
			uniform mat4 project;
			void main() {
				vec4 pos = project * vec4(pos, -1.0, 1.0);
				pos.xyz /= pos.w;
				pos.y *= -1.0;
				gl_Position = pos;
			})";

		static const char* shader_frag = R"(#version 140
			out vec4 frag_color;
			void main() {
				frag_color = vec4(0.0, 0.0, 0.0, 1.0);
			})";

		GLuint vert = glCreateShader(GL_VERTEX_SHADER);
		glShaderSource(vert, 1, &shader_vert, NULL);
		glCompileShader(vert);
		check_shader(vert, "OpenXR hidden area mask vertex shader", false);

		GLuint frag = glCreateShader(GL_FRAGMENT_SHADER);
		glShaderSource(frag, 1, &shader_frag, NULL);
		glCompileShader(frag);
		check_shader(frag, "OpenXR hidden area mask fragment shader", false);

		m_hidden_area_mask_program = glCreateProgram();
		glAttachShader(m_hidden_area_mask_program, vert);
		glAttachShader(m_hidden_area_mask_program, frag);
		glLinkProgram(m_hidden_area_mask_program);
		check_shader(m_hidden_area_mask_program, "OpenXR hidden area mask shader program", true);

		glDeleteShader(vert);
		glDeleteShader(frag);
	}
}

void OpenXRHMD::session_state_change(XrSessionState state, EControlFlow& flow) {
	//tlog::info() << fmt::format("New session state {}", XrEnumStr(state));
	switch (state) {
		case XR_SESSION_STATE_READY: {
			XrSessionBeginInfo sessionBeginInfo {XR_TYPE_SESSION_BEGIN_INFO};
			sessionBeginInfo.primaryViewConfigurationType = m_view_configuration_type;
			XR_CHECK_THROW(xrBeginSession(m_session, &sessionBeginInfo));
			break;
		}
		case XR_SESSION_STATE_STOPPING: {
			XR_CHECK_THROW(xrEndSession(m_session));
			break;
		}
		case XR_SESSION_STATE_EXITING: {
			flow = EControlFlow::Quit;
			break;
		}
		case XR_SESSION_STATE_LOSS_PENDING: {
			flow = EControlFlow::Restart;
			break;
		}
		default: {
			break;
		}
	}
}

OpenXRHMD::EControlFlow OpenXRHMD::poll_events() {
	bool more = true;
	EControlFlow flow = EControlFlow::Continue;
	while (more) {
		// poll events
		XrEventDataBuffer event {XR_TYPE_EVENT_DATA_BUFFER, nullptr};
		XrResult result = xrPollEvent(m_instance, &event);

		if (XR_FAILED(result)) {
			tlog::error() << "xrPollEvent failed";
		} else if (XR_SUCCESS == result) {
			switch (event.type) {
				case XR_TYPE_EVENT_DATA_SESSION_STATE_CHANGED: {
					const XrEventDataSessionStateChanged& e = *reinterpret_cast<XrEventDataSessionStateChanged*>(&event);
					//tlog::info() << "Session state change";
					//tlog::info() << fmt::format("\t from {}\t   to {}", XrEnumStr(m_session_state), XrEnumStr(e.state));
					//tlog::info() << fmt::format("\t session {}, time {}", fmt::ptr(e.session), e.time);
					m_session_state = e.state;
					session_state_change(e.state, flow);
					break;
				}

				case XR_TYPE_EVENT_DATA_INSTANCE_LOSS_PENDING: {
					flow = EControlFlow::Restart;
					break;
				}

				case XR_TYPE_EVENT_DATA_VISIBILITY_MASK_CHANGED_KHR: {
					m_hidden_area_masks.clear();
					break;
				}

				case XR_TYPE_EVENT_DATA_INTERACTION_PROFILE_CHANGED: {
					break; // Can ignore
				}

				default: {
					tlog::info() << fmt::format("Unhandled event type {}", XrEnumStr(event.type));
					break;
				}
			}
		} else if (XR_EVENT_UNAVAILABLE == result) {
			more = false;
		}
	}
	return flow;
}

__global__ void read_hidden_area_mask_kernel(const Vector2i resolution, hipSurfaceObject_t surface, uint8_t* __restrict__ mask) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;
	surf2Dread(&mask[idx], surface, x, y);
}

std::shared_ptr<Buffer2D<uint8_t>> OpenXRHMD::rasterize_hidden_area_mask(uint32_t view_index, const XrCompositionLayerProjectionView& view) {
	if (!m_supports_hidden_area_mask) {
		return {};
	}

	PFN_xrGetVisibilityMaskKHR xrGetVisibilityMaskKHR = nullptr;
	XR_CHECK_THROW(xrGetInstanceProcAddr(
		m_instance,
		"xrGetVisibilityMaskKHR",
		reinterpret_cast<PFN_xrVoidFunction*>(&xrGetVisibilityMaskKHR)
	));

	XrVisibilityMaskKHR visibility_mask{XR_TYPE_VISIBILITY_MASK_KHR};
	XR_CHECK_THROW(xrGetVisibilityMaskKHR(m_session, m_view_configuration_type, view_index, XR_VISIBILITY_MASK_TYPE_HIDDEN_TRIANGLE_MESH_KHR, &visibility_mask));

	if (visibility_mask.vertexCountOutput == 0 || visibility_mask.indexCountOutput == 0) {
		return nullptr;
	}

	std::vector<XrVector2f> vertices(visibility_mask.vertexCountOutput);
	std::vector<uint32_t> indices(visibility_mask.indexCountOutput);

	visibility_mask.vertices = vertices.data();
	visibility_mask.indices = indices.data();

	visibility_mask.vertexCapacityInput = visibility_mask.vertexCountOutput;
	visibility_mask.indexCapacityInput = visibility_mask.indexCountOutput;

	XR_CHECK_THROW(xrGetVisibilityMaskKHR(m_session, m_view_configuration_type, view_index, XR_VISIBILITY_MASK_TYPE_HIDDEN_TRIANGLE_MESH_KHR, &visibility_mask));

	CUDA_CHECK_THROW(hipDeviceSynchronize());

	Vector2i size = {view.subImage.imageRect.extent.width, view.subImage.imageRect.extent.height};

	bool tex = glIsEnabled(GL_TEXTURE_2D);
	bool depth = glIsEnabled(GL_DEPTH_TEST);
	bool cull = glIsEnabled(GL_CULL_FACE);
	GLint previous_texture_id;
	glGetIntegerv(GL_TEXTURE_BINDING_2D, &previous_texture_id);

	if (!tex) glEnable(GL_TEXTURE_2D);
	if (depth) glDisable(GL_DEPTH_TEST);
	if (cull) glDisable(GL_CULL_FACE);

	// Generate texture to hold hidden area mask. Single channel, value of 1 means visible and 0 means masked away
	ngp::GLTexture mask_texture;
	mask_texture.resize(size, 1, true);
	glBindTexture(GL_TEXTURE_2D, mask_texture.texture());
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

	GLuint framebuffer = 0;
	glGenFramebuffers(1, &framebuffer);
	glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
	glFramebufferTexture(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, mask_texture.texture(), 0);

	GLenum draw_buffers[1] = {GL_COLOR_ATTACHMENT0};
	glDrawBuffers(1, draw_buffers);

	glViewport(0, 0, size.x(), size.y());

	// Draw hidden area mask
	GLuint vao;
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);

	GLuint vertex_buffer;
	glGenBuffers(1, &vertex_buffer);
	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, vertex_buffer);
	glBufferData(GL_ARRAY_BUFFER, sizeof(XrVector2f) * vertices.size(), vertices.data(), GL_STATIC_DRAW);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, (void*)0);

	GLuint index_buffer;
	glGenBuffers(1, &index_buffer);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, index_buffer);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(uint32_t) * indices.size(), indices.data(), GL_STATIC_DRAW);

	glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT);
	glUseProgram(m_hidden_area_mask_program);

	XrMatrix4x4f proj;
	XrMatrix4x4f_CreateProjectionFov(&proj, GRAPHICS_OPENGL, view.fov, 1.0f / 128.0f, 128.0f);

	GLuint project_id = glGetUniformLocation(m_hidden_area_mask_program, "project");
	glUniformMatrix4fv(project_id, 1, GL_FALSE, &proj.m[0]);

	glDrawElements(GL_TRIANGLES, indices.size(), GL_UNSIGNED_INT, (void*)0);
	glFinish();

	glDisableVertexAttribArray(0);
	glDeleteBuffers(1, &vertex_buffer);
	glDeleteBuffers(1, &index_buffer);
	glDeleteVertexArrays(1, &vao);
	glDeleteFramebuffers(1, &framebuffer);

	glBindVertexArray(0);
	glUseProgram(0);

	// restore old state
	if (!tex) glDisable(GL_TEXTURE_2D);
	if (depth) glEnable(GL_DEPTH_TEST);
	if (cull) glEnable(GL_CULL_FACE);
	glBindTexture(GL_TEXTURE_2D, previous_texture_id);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);

	std::shared_ptr<Buffer2D<uint8_t>> mask = std::make_shared<Buffer2D<uint8_t>>(size);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)size.x(), threads.x), div_round_up((uint32_t)size.y(), threads.y), 1 };

	read_hidden_area_mask_kernel<<<blocks, threads>>>(size, mask_texture.surface(), mask->data());
	CUDA_CHECK_THROW(hipDeviceSynchronize());

	return mask;
}

Matrix<float, 3, 4> convert_xr_matrix_to_eigen(const XrMatrix4x4f& m) {
	Matrix<float, 3, 4> out;

	for (size_t i = 0; i < 3; ++i) {
		for (size_t j = 0; j < 4; ++j) {
			out(i, j) = m.m[i + j * 4];
		}
	}

	// Flip Y and Z axes to match NGP conventions
	out(0, 1) *= -1.f;
	out(1, 0) *= -1.f;

	out(0, 2) *= -1.f;
	out(2, 0) *= -1.f;

	out(1, 3) *= -1.f;
	out(2, 3) *= -1.f;

	return out;
}

Matrix<float, 3, 4> convert_xr_pose_to_eigen(const XrPosef& pose) {
	XrMatrix4x4f matrix;
	XrVector3f unit_scale{1.0f, 1.0f, 1.0f};
	XrMatrix4x4f_CreateTranslationRotationScale(&matrix, &pose.position, &pose.orientation, &unit_scale);
	return convert_xr_matrix_to_eigen(matrix);
}

OpenXRHMD::FrameInfoPtr OpenXRHMD::begin_frame() {
	XrFrameWaitInfo frame_wait_info{XR_TYPE_FRAME_WAIT_INFO};
	XR_CHECK_THROW(xrWaitFrame(m_session, &frame_wait_info, &m_frame_state));

	XrFrameBeginInfo frame_begin_info{XR_TYPE_FRAME_BEGIN_INFO};
	XR_CHECK_THROW(xrBeginFrame(m_session, &frame_begin_info));

	if (!m_frame_state.shouldRender) {
		return std::make_shared<FrameInfo>();
	}

	uint32_t num_views = (uint32_t)m_swapchains.size();
	// TODO assert m_view_configuration_views.size() == m_swapchains.size()

	// locate views
	std::vector<XrView> views(num_views, {XR_TYPE_VIEW});

	XrViewState viewState{XR_TYPE_VIEW_STATE};

	XrViewLocateInfo view_locate_info{XR_TYPE_VIEW_LOCATE_INFO};
	view_locate_info.viewConfigurationType = m_view_configuration_type;
	view_locate_info.displayTime = m_frame_state.predictedDisplayTime;
	view_locate_info.space = m_space;

	XR_CHECK_THROW(xrLocateViews(m_session, &view_locate_info, &viewState, uint32_t(views.size()), &num_views, views.data()));

	if (!(viewState.viewStateFlags & XR_VIEW_STATE_POSITION_VALID_BIT) || !(viewState.viewStateFlags & XR_VIEW_STATE_ORIENTATION_VALID_BIT)) {
		return std::make_shared<FrameInfo>();
	}

	m_hidden_area_masks.resize(num_views);

	// Fill frame information
	if (!m_previous_frame_info) {
		m_previous_frame_info = std::make_shared<FrameInfo>();
	}

	FrameInfoPtr frame_info = std::make_shared<FrameInfo>(*m_previous_frame_info);
	frame_info->views.resize(m_swapchains.size());

	for (size_t i = 0; i < m_swapchains.size(); ++i) {
		const auto& sc = m_swapchains[i];

		XrSwapchainImageAcquireInfo image_acquire_info{XR_TYPE_SWAPCHAIN_IMAGE_ACQUIRE_INFO};
		XrSwapchainImageWaitInfo image_wait_info{XR_TYPE_SWAPCHAIN_IMAGE_WAIT_INFO, nullptr, XR_INFINITE_DURATION};

		uint32_t image_index;
		XR_CHECK_THROW(xrAcquireSwapchainImage(sc.handle, &image_acquire_info, &image_index));
		XR_CHECK_THROW(xrWaitSwapchainImage(sc.handle, &image_wait_info));

		FrameInfo::View& v = frame_info->views[i];
		v.framebuffer = sc.framebuffers_gl[image_index];
		v.view.pose = views[i].pose;
		v.view.fov = views[i].fov;
		v.view.subImage.imageRect = XrRect2Di{{0, 0}, {sc.width, sc.height}};
		v.view.subImage.imageArrayIndex = 0;
		v.view.subImage.swapchain = sc.handle;

		glBindFramebuffer(GL_FRAMEBUFFER, sc.framebuffers_gl[image_index]);
		glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, sc.images_gl.at(image_index).image, 0);

		if (sc.depth_handle != XR_NULL_HANDLE) {
			uint32_t depth_image_index;
			XR_CHECK_THROW(xrAcquireSwapchainImage(sc.depth_handle, &image_acquire_info, &depth_image_index));
			XR_CHECK_THROW(xrWaitSwapchainImage(sc.depth_handle, &image_wait_info));

			glFramebufferTexture2D(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_TEXTURE_2D, sc.depth_images_gl.at(depth_image_index).image, 0);

			v.depth_info.subImage.imageRect = XrRect2Di{{0, 0}, {sc.width, sc.height}};
			v.depth_info.subImage.imageArrayIndex = 0;
			v.depth_info.subImage.swapchain = sc.depth_handle;
			v.depth_info.minDepth = 0.0f;
			v.depth_info.maxDepth = 1.0f;

			// To be overwritten with actual near and far planes by end_frame
			v.depth_info.nearZ = 1.0f / 128.0f;
			v.depth_info.farZ = 128.0f;
		}

		glBindFramebuffer(GL_FRAMEBUFFER, 0);

		if (!m_hidden_area_masks.at(i)) {
			m_hidden_area_masks.at(i) = rasterize_hidden_area_mask(i, v.view);
		}

		v.hidden_area_mask = m_hidden_area_masks.at(i);
		v.pose = convert_xr_pose_to_eigen(v.view.pose);
	}

	XrActiveActionSet active_action_set{m_action_set, XR_NULL_PATH};
	XrActionsSyncInfo sync_info{XR_TYPE_ACTIONS_SYNC_INFO};
	sync_info.countActiveActionSets = 1;
	sync_info.activeActionSets = &active_action_set;
	XR_CHECK_THROW(xrSyncActions(m_session, &sync_info));

	for (size_t i = 0; i < 2; ++i) {
		// Hand pose
		{
			XrActionStatePose pose_state{XR_TYPE_ACTION_STATE_POSE};
			XrActionStateGetInfo get_info{XR_TYPE_ACTION_STATE_GET_INFO};
			get_info.action = m_pose_action;
			get_info.subactionPath = m_hand_paths[i];
			XR_CHECK_THROW(xrGetActionStatePose(m_session, &get_info, &pose_state));

			frame_info->hands[i].pose_active = pose_state.isActive;
			if (frame_info->hands[i].pose_active) {
				XrSpaceLocation space_location{XR_TYPE_SPACE_LOCATION};
				XR_CHECK_THROW(xrLocateSpace(m_hand_spaces[i], m_space, m_frame_state.predictedDisplayTime, &space_location));
				frame_info->hands[i].pose = convert_xr_pose_to_eigen(space_location.pose);
			}
		}

		// Stick
		{
			XrActionStateVector2f thumbstick_state{XR_TYPE_ACTION_STATE_VECTOR2F};
			XrActionStateGetInfo get_info{XR_TYPE_ACTION_STATE_GET_INFO};
			get_info.action = m_thumbstick_actions[i];
			XR_CHECK_THROW(xrGetActionStateVector2f(m_session, &get_info, &thumbstick_state));

			if (thumbstick_state.isActive) {
				frame_info->hands[i].thumbstick.x() = thumbstick_state.currentState.x;
				frame_info->hands[i].thumbstick.y() = thumbstick_state.currentState.y;
			} else {
				frame_info->hands[i].thumbstick = Vector2f::Zero();
			}
		}

		// Press
		{
			XrActionStateBoolean press_state{XR_TYPE_ACTION_STATE_BOOLEAN};
			XrActionStateGetInfo get_info{XR_TYPE_ACTION_STATE_GET_INFO};
			get_info.action = m_press_action;
			get_info.subactionPath = m_hand_paths[i];
			XR_CHECK_THROW(xrGetActionStateBoolean(m_session, &get_info, &press_state));

			if (press_state.isActive) {
				frame_info->hands[i].pressing = press_state.currentState;
			} else {
				frame_info->hands[i].pressing = 0.0f;
			}
		}

		// Grab
		{
			XrActionStateFloat grab_state{XR_TYPE_ACTION_STATE_FLOAT};
			XrActionStateGetInfo get_info{XR_TYPE_ACTION_STATE_GET_INFO};
			get_info.action = m_grab_action;
			get_info.subactionPath = m_hand_paths[i];
			XR_CHECK_THROW(xrGetActionStateFloat(m_session, &get_info, &grab_state));

			if (grab_state.isActive) {
				frame_info->hands[i].grab_strength = grab_state.currentState;
			} else {
				frame_info->hands[i].grab_strength = 0.0f;
			}

			bool was_grabbing = frame_info->hands[i].grabbing;
			frame_info->hands[i].grabbing = frame_info->hands[i].grab_strength >= 0.5f;

			if (frame_info->hands[i].grabbing) {
				frame_info->hands[i].prev_grab_pos = was_grabbing ? frame_info->hands[i].grab_pos : frame_info->hands[i].pose.col(3);
				frame_info->hands[i].grab_pos = frame_info->hands[i].pose.col(3);
			}
		}
	}

	m_previous_frame_info = frame_info;
	return frame_info;
}

void OpenXRHMD::end_frame(FrameInfoPtr frame_info, float znear, float zfar, bool submit_depth) {
	std::vector<XrCompositionLayerProjectionView> layer_projection_views(frame_info->views.size());
	for (size_t i = 0; i < layer_projection_views.size(); ++i) {
		auto& v = frame_info->views[i];
		auto& view = layer_projection_views[i];

		view = v.view;

		// release swapchain image
		XrSwapchainImageReleaseInfo release_info{XR_TYPE_SWAPCHAIN_IMAGE_RELEASE_INFO};
		XR_CHECK_THROW(xrReleaseSwapchainImage(v.view.subImage.swapchain, &release_info));

		if (v.depth_info.subImage.swapchain != XR_NULL_HANDLE) {
			XR_CHECK_THROW(xrReleaseSwapchainImage(v.depth_info.subImage.swapchain, &release_info));
			v.depth_info.nearZ = znear;
			v.depth_info.farZ = zfar;

			// Submitting the depth buffer to the runtime for reprojection is optional,
			// because, while depth-based reprojection can make the experience smoother,
			// it also results in distortion around geometric edges. Many users prefer
			// a more stuttery experience without this distortion.
			if (submit_depth) {
				view.next = &v.depth_info;
			}
		}
	}

	XrCompositionLayerProjection layer{XR_TYPE_COMPOSITION_LAYER_PROJECTION};
	layer.space = m_space;
	if (m_environment_blend_mode != EEnvironmentBlendMode::Opaque) {
		layer.layerFlags = XR_COMPOSITION_LAYER_BLEND_TEXTURE_SOURCE_ALPHA_BIT;
	}

	layer.viewCount = uint32_t(layer_projection_views.size());
	layer.views = layer_projection_views.data();

	std::vector<XrCompositionLayerBaseHeader*> layers;
	if (layer.viewCount) {
		layers.push_back(reinterpret_cast<XrCompositionLayerBaseHeader*>(&layer));
	}

	XrFrameEndInfo frame_end_info{XR_TYPE_FRAME_END_INFO};
	frame_end_info.displayTime = m_frame_state.predictedDisplayTime;
	frame_end_info.environmentBlendMode = (XrEnvironmentBlendMode)m_environment_blend_mode;
	frame_end_info.layerCount = (uint32_t)layers.size();
	frame_end_info.layers = layers.data();
	XR_CHECK_THROW(xrEndFrame(m_session, &frame_end_info));
}

NGP_NAMESPACE_END

#ifdef __GNUC__
#pragma GCC diagnostic pop
#endif
